#include "hip/hip_runtime.h"
/**
 * linear combination for 2-D matrix
 * Copyright 2018 (C) Johann Lee <me@qinka.pro>
 */


#ifndef _LINEAR_COMBINATION_C_
#define _LINEAR_COMBINATION_C_

#include <linear-combination.h>
#include <math.h>
#include <stdio.h>
#include <stdint.h>


__global__
void linearCombinKernel(float coe1, uint8_t* m1, float coe2, uint8_t* m2, int row, int col, uint8_t* m3) {
  int idxX = blockIdx.x * blockDim.x + threadIdx.x;
  int idxY = blockIdx.y * blockDim.y + threadIdx.y;
  int stdX = blockDim.x * gridDim.x;
  int stdY = blockDim.y * gridDim.y;
  for(int i = idxX; i < row; i += stdX)
    for(int j = idxY; j < col; j += stdY) {
      float tmp = coe1 * m1[i*col + j] + coe2 * m2[i*col + j];
      m3[i*col+j] = (uint8_t)(fmaxf(fminf(tmp,255),0));
    }
}


int linear_combination(float coe1, uint8_t* m1, float coe2, uint8_t* m2, int row, int col, uint8_t* m3) {
  uint8_t* dm1 = 0;
  uint8_t* dm2 = 0;
  uint8_t* dm3 = 0;
  hipError_t rtCode;
  dim3 blocksize;
  dim3 threadsPerBlock;

  // select codes
  rtCode = hipSetDevice(0);
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    goto Error;
  }

  // get prop
  hipDeviceProp_t prop;
  rtCode != hipGetDeviceProperties(&prop,0);
  if (rtCode != hipSuccess) {
    fprintf(stderr, "Fail to get the device infos");
    goto Error;
  }

  // malloc
  rtCode = hipMalloc((void**)&dm1, col * row  * sizeof(uint8_t));
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!(m1)");
    goto Error;
  }
  rtCode = hipMalloc((void**)&dm2, col * row  * sizeof(uint8_t));
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!(m2)");
    goto Error;
  }
  rtCode = hipMalloc((void**)&dm3, col * row  * sizeof(uint8_t));
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!(m3)");
    goto Error;
  }

  // copy
  rtCode = hipMemcpy(dm1, m1, col * row * sizeof(uint8_t), hipMemcpyHostToDevice);
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!(m1)");
    goto Error;
  }
  rtCode = hipMemcpy(dm2, m2, col * row * sizeof(uint8_t), hipMemcpyHostToDevice);
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!(m2)");
    goto Error;
  }

  // run
  blocksize = dim3(prop.maxGridSize[0],prop.maxGridSize[1]);
  threadsPerBlock = dim3(prop.maxThreadsDim[0],prop.maxThreadsDim[1]);
  linearCombinKernel<<<blocksize,threadsPerBlock>>>(coe1,m1,coe2,m2,row,col,m3);

  // check error
  rtCode = hipGetLastError();
  if (rtCode != hipSuccess) {
    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(rtCode));
    goto Error;
  }

  // synchronize
  rtCode = hipDeviceSynchronize();
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", rtCode);
    goto Error;
  }

  // copy
  rtCode = hipMemcpy(m3, dm3, row * col * sizeof(int), hipMemcpyDeviceToHost);
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!(m3)");
    goto Error;
  }

Error:
    hipFree(dm1);
    hipFree(dm2);
    hipFree(dm3);
    return rtCode;
}


#endif // _LINEAR_COMBINATION_C_
