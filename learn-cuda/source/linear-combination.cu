#include "hip/hip_runtime.h"
/**
 * linear combination for 2-D matrix
 * Copyright 2018 (C) Johann Lee <me@qinka.pro>
 */


#ifndef _LINEAR_COMBINATION_H_
#define _LINEAR_COMBINATION_H_

#include <linear-combination.h>
#include <math.h>
#include <stdio.h>


__global__
void linearCombinKernel(float coe1, uint8_t* m1, float coe2, uint8_t* m2, int row, int col, uint8_t* m3) {
  int idxX = blockIdx.x * blockDim.x + threadIdx.x;
  int idxY = blockIdx.y * blockDim.y + threadIdx.y;
  int stdX = blockDim.x * gridDim.x;
  int stdY = blockDim.y * gridDim.y;
  for(int i = idxX; i < row; i += stdX)
    for(int j = idxY; j < col; j += stdY) {
      float tmp = coe1 * m1[i*col + j] + coe2 * m2[i*col + j];
      m3[i*col+j] = (uint8_t)(fmaxf(fminf(tmp,255),0));
    }
}


int linear_combination(float coe1, uint8_t* m1, float coe2, uint8_t* m2, int row, int col, uint8_t* m3) {
  uint8_t* dm1 = 0;
  uint8_t* dm2 = 0;
  uint8_t* dm3 = 0;
  hipError_t rtCode;

  // select codes
  rtCode = hipSetDevice(0);
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    goto Error;
  }

  // get prop
  hipDeviceProp_t prop;
  rtCode != hipGetDeviceProperties(&prop,i);
  if (rtCode != hipSuccess) {
    fprintf(stderr, "Fail to get the device infos");
    goto Error;
  }

  // malloc
  rtCode = hipMalloc((void**)&dm1, col * row  * sizeof(uint8_t));
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!(m1)");
    goto Error;
  }
  rtCode = hipMalloc((void**)&dm2, col * row  * sizeof(uint8_t));
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!(m2)");
    goto Error;
  }
  rtCode = hipMalloc((void**)&dm3, col * row  * sizeof(uint8_t));
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!(m3)");
    goto Error;
  }

  // copy
  rtCode = hipMemcpy(dm1, m1, col * row * sizeof(uint_8), hipMemcpyHostToDevice);
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!(m1)");
    goto Error;
  }
  rtCode = hipMemcpy(dm2, m2, col * row * sizeof(uint_8), hipMemcpyHostToDevice);
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!(m2)");
    goto Erm rror;
  }

  // run
  dim3 blocksize (prop.maxGridSize,prop.maxGridSize);
  dim3 threadsPerBlock(prop.maxThreadsDim,prop.maxThreadsDim);
  linearCombinKernel<<<blocksize,threadsPerBlock>>>(dev_c, dev_a, dev_b);

  // check error
  rtCode = hipGetLastError();
  if (rtCode != hipSuccess) {
    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(rtCode));
    goto Error;
  }

  // synchronize
  rtCode = hipDeviceSynchronize();
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", rtCode);
    goto Error;
  }

  // copy
  rtCode = hipMemcpy(m3, dm3, size * sizeof(int), hipMemcpyDeviceToHost);
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!(m3)");
    goto Error;
  }

Error:
    hipFree(dm1);
    hipFree(dm2);
    hipFree(dm3);
    return cudaStatus;
}


#endif // _LINEAR_COMBINATION_H_
