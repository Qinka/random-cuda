#include "hip/hip_runtime.h"
/**
 * linear combination for 2-D matrix
 * Copyright 2018 (C) Johann Lee <me@qinka.pro>
 */


#ifndef _LINEAR_COMBINATION_C_
#define _LINEAR_COMBINATION_C_

#include <linear-combination.h>
#include <math.h>
#include <stdio.h>
#include <stdint.h>


__global__
void linearCombinKernel(float coe1, uint8_t* m1, float coe2, uint8_t* m2, int size, uint8_t* m3) {
  int idxX = blockIdx.x * blockDim.x + threadIdx.x;
  int stdX = blockDim.x * gridDim.x;
  for(int i = idxX; i < size; i += stdX) {
    float tmp = coe1 * m1[i] + coe2 * m2[i];
    m3[i] = (uint8_t)(fmaxf(fminf(tmp,255),0));
  }
}


int linear_combination(float coe1, uint8_t* m1, float coe2, uint8_t* m2, int _size, uint8_t* m3) {
  uint8_t* dm1 = 0;
  uint8_t* dm2 = 0;
  uint8_t* dm3 = 0;
  hipError_t rtCode;
  int blocksize;
  int threadsPerBlock;
  size_t size = _size * sizeof(uint8_t);
  int bsX = 0;

  // select codes
  /*rtCode = hipSetDevice(0);
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    goto Error;
    }*/

  // get prop
  hipDeviceProp_t prop;
  rtCode = hipGetDeviceProperties(&prop,0);
  if (rtCode != hipSuccess) {
    fprintf(stderr, "Fail to get the device infos");
    goto Error;
  }

  // malloc
  rtCode = hipMalloc((void**)&dm1, size);
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!(m1)");
    goto Error;
  }
  rtCode = hipMalloc((void**)&dm2,size);
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!(m2)");
    goto Error;
  }
  rtCode = hipMalloc((void**)&dm3, size);
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!(m3)");
    goto Error;
  }

  // copy
  rtCode = hipMemcpy(dm1, m1, size, hipMemcpyHostToDevice);
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!(m1)");
    goto Error;
  }
  rtCode = hipMemcpy(dm2, m2, size, hipMemcpyHostToDevice);
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!(m2)");
    goto Error;
  }

  // run

  bsX = (size / prop.maxThreadsPerBlock) + 1;
  blocksize = min((size_t)prop.maxGridSize[0],bsX);
  threadsPerBlock = min(prop.maxThreadsPerBlock,size);
  linearCombinKernel<<<blocksize,threadsPerBlock>>>(coe1,dm1,coe2,dm2,_size,dm3);

  // check error
  rtCode = hipGetLastError();
  if (rtCode != hipSuccess) {
    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(rtCode));
    goto Error;
  }

  // synchronize
  rtCode = hipDeviceSynchronize();
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", rtCode);
    goto Error;
  }

  // copy
  rtCode = hipMemcpy(m3, dm3, size, hipMemcpyDeviceToHost);
  if (rtCode != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!(m3),%d",rtCode);
    goto Error;
  }

Error:
    hipFree(dm1);
    hipFree(dm2);
    hipFree(dm3);
    return rtCode;
}


#endif // _LINEAR_COMBINATION_C_
